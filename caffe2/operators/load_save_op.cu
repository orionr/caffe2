#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/load_save_op.h"

namespace caffe2 {
namespace {
REGISTER_HIP_OPERATOR(LoadFloatTensor, LoadFloatTensorOp<HIPContext>);
REGISTER_HIP_OPERATOR(SaveFloatTensor, SaveFloatTensorOp<HIPContext>);
}  // namespace
}  // namespace caffe2
