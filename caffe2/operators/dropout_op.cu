#include "hip/hip_runtime.h"
#include "caffe2/operators/dropout_op.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {
__global__ void DropoutKernel(const int N, const float ratio,
                              const float* Xdata, float* Ydata,
                              bool* maskdata) {
  const float scale = 1. / (1. - ratio);
  CUDA_1D_KERNEL_LOOP(i, N) {
    maskdata[i] = (Ydata[i] > ratio);
    Ydata[i] = Xdata[i] * scale * maskdata[i];
  }
}
}  // namespace

template <>
bool DropoutOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  auto* mask = OperatorBase::Output<Tensor<bool, HIPContext> >(1);
  Y->Reshape(X.dims());
  mask->Reshape(X.dims());
  DCHECK_GT(X.size(), 0);
  // We do a simple trick here: since hiprand cannot generate random
  // boolean numbers, we will generate into dY and write the result to
  // mask.
  float* Ydata = Y->mutable_data();
  CURAND_CHECK(hiprandGenerateUniform(
      device_context_.curand_generator(), Ydata, X.size()));
  DropoutKernel<<<CAFFE_GET_BLOCKS(X.size()), CAFFE_CUDA_NUM_THREADS,
                  0, device_context_.hip_stream()>>>(
      X.size(), ratio_, X.data(), Ydata, mask->mutable_data());
  return true;
}

namespace {
__global__ void DropoutGradientKernel(const int N, const float* dYdata,
                                      const bool* maskdata, float* dXdata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dXdata[i] = dYdata[i] * maskdata[i];
  }
}
}  // namespace

template <>
bool DropoutGradientOp<float, HIPContext>::RunOnDevice() {
  auto& dY = Input(0);
  auto& mask =
      OperatorBase::Input<Tensor<bool, HIPContext> >(1);
  auto* dX = Output(0);
  DCHECK_GT(dY.size(), 0);
  DCHECK_EQ(dY.size(), mask.size());
  dX->Reshape(dY.dims());
  DropoutGradientKernel<<<CAFFE_GET_BLOCKS(dY.size()),
                          CAFFE_CUDA_NUM_THREADS,
                          0, device_context_.hip_stream()>>>(
      dY.size(), dY.data(), mask.data(), dX->mutable_data());
  return true;
}


namespace {
REGISTER_HIP_OPERATOR(Dropout, DropoutOp<float, HIPContext>)
REGISTER_HIP_OPERATOR(DropoutGrad, DropoutGradientOp<float, HIPContext>)
}  // namespace
}  // namespace caffe2
