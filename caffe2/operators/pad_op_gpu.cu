#include <algorithm>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/pad_op.h"

namespace caffe2 {

template <>
bool PadImageOp<float, HIPContext>::RunOnDeviceWithOrderNCHW() {
  CAFFE_NOT_IMPLEMENTED;
}

template<>
bool PadImageOp<float, HIPContext>::RunOnDeviceWithOrderNHWC() {
  CAFFE_NOT_IMPLEMENTED;
}

template<>
bool PadImageGradientOp<float, HIPContext>::RunOnDeviceWithOrderNCHW() {
  CAFFE_NOT_IMPLEMENTED;
}

template<>
bool PadImageGradientOp<float, HIPContext>::RunOnDeviceWithOrderNHWC() {
  CAFFE_NOT_IMPLEMENTED;
}


REGISTER_HIP_OPERATOR(PadImage, PadImageOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(PadImageGradient,
                       PadImageGradientOp<float, HIPContext>);
}  // namespace caffe2
