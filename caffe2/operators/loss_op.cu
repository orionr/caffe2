#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/loss_op.h"

namespace caffe2 {

namespace {
template <typename T>
__global__ void ALGKernel(const int N, const T* dY, T* dX) {
  const T value = (*dY) / N; 
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = value;
  }
}
}  // namespace

class AveragedLossGradientGPUSpecialization final
    : public Operator<HIPContext> {
 public:
  AveragedLossGradientGPUSpecialization(
      const OperatorDef& operator_def, Workspace* ws)
      : Operator<HIPContext>(operator_def, ws) {}
  ~AveragedLossGradientGPUSpecialization() {}
  USE_OPERATOR_FUNCTIONS(HIPContext);

  bool RunOnDevice() override {
    auto& X = Input(0);
    auto& dY = Input(1);
    DCHECK_EQ(dY.size(), 1);
    auto* dX = Output(0);
    dX->ResizeLike(X);
    ALGKernel<float><<<CAFFE_GET_BLOCKS(X.size()), CAFFE_CUDA_NUM_THREADS,
                       0, context_.hip_stream()>>>(
        X.size(), dY.data<float>(), dX->mutable_data<float>());
    return true;
  }
};

namespace {
REGISTER_HIP_OPERATOR(AveragedLoss, AveragedLoss<float, HIPContext>);
REGISTER_HIP_OPERATOR(AveragedLossGradient,
                       AveragedLossGradientGPUSpecialization);
}  // namespace
}  // namespace caffe2

