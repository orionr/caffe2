#include "hip/hip_runtime.h"
#include "momentum_sgd_op.h"
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

__global__ void MomentumSGDUpdate(
    int N,
    const float* g,
    const float* m,
    float* ng,
    float* nm,
    const float* lr,
    float momentum,
    bool nesterov) {
  if (!nesterov) {
    CUDA_1D_KERNEL_LOOP(i, N) {
      const float adjusted_gradient = lr[0] * g[i] + momentum * m[i];
      nm[i] = adjusted_gradient;
      ng[i] = adjusted_gradient;
    }
  } else {
    CUDA_1D_KERNEL_LOOP(i, N) {
      const float mi = m[i];
      const float mi_new = momentum * mi + lr[0] * g[i];
      nm[i] = mi_new;
      ng[i] = (1 + momentum) * mi_new - momentum * mi;
    }
  }
}

template<>
void momentum_sgd_update<HIPContext>(
    int N,
    const float* g,
    const float* m,
    float* ng,
    float* nm,
    const float* lr,
    float momentum,
    bool nesterov,
    HIPContext* context) {
  MomentumSGDUpdate<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      N, g, m, ng, nm, lr, momentum, nesterov);
}

namespace {
REGISTER_HIP_OPERATOR(MomentumSGD, MomentumSGDOp<float, HIPContext>);
}

}
