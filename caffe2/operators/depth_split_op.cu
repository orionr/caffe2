#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/depth_split_op.h"

namespace caffe2 {
namespace {
REGISTER_HIP_OPERATOR(DepthSplit, DepthSplitOp<float, HIPContext>)
REGISTER_HIP_OPERATOR(DepthConcat, DepthConcatOp<float, HIPContext>)
}  // namespace
}  // namespace caffe2

