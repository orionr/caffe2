#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/core/operator.h"

namespace caffe2 {

namespace {
__global__ void FloatToHalfKernel(const int N, const float* X, float16* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __float2half_rn(X[i]);
  }
}

__global__ void HalfToFloatKernel(const int N, const float16* X, float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __half2float(X[i]);
  }
}
}

class FloatToHalfCUDA : public Operator<HIPContext> {
 public:
  FloatToHalfCUDA(const OperatorDef& def, Workspace* ws)
      : Operator<HIPContext>(def, ws) {}
  ~FloatToHalfCUDA() {}

  bool RunOnDevice() {
    auto& X = Input(0);
    auto* Y = Output(0);
    Y->ReshapeLike(X);
    FloatToHalfKernel<<<CAFFE_GET_BLOCKS(X.size()), CAFFE_CUDA_NUM_THREADS,
                        0, device_context_.hip_stream()>>>(
      X.size(), X.data<float>(), Y->mutable_data<float16>());
  return true;
    return true;
  }

  INPUT_OUTPUT_STATS(1, 1, 1, 1);
  DISABLE_COPY_AND_ASSIGN(FloatToHalfCUDA);
};

class HalfToFloatCUDA : public Operator<HIPContext> {
 public:
  HalfToFloatCUDA(const OperatorDef& def, Workspace* ws)
      : Operator<HIPContext>(def, ws) {}
  ~HalfToFloatCUDA() {}

  bool RunOnDevice() {
    auto& X = Input(0);
    auto* Y = Output(0);
    Y->ReshapeLike(X);
    HalfToFloatKernel<<<CAFFE_GET_BLOCKS(X.size()), CAFFE_CUDA_NUM_THREADS,
                        0, device_context_.hip_stream()>>>(
      X.size(), X.data<float16>(), Y->mutable_data<float>());
  return true;
    return true;
  }

  INPUT_OUTPUT_STATS(1, 1, 1, 1);
  DISABLE_COPY_AND_ASSIGN(HalfToFloatCUDA);
};

REGISTER_HIP_OPERATOR(FloatToHalf, FloatToHalfCUDA)
REGISTER_HIP_OPERATOR(HalfToFloat, HalfToFloatCUDA)
}  // namespace caffe2