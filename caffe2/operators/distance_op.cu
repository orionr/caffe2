#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/distance_op.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void SquaredL2DistanceKernel(
    const int N, const int D, const T* X, const T* Y, T* distance) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  for (int i = blockIdx.x; i < N; i += gridDim.x) {
    float dist = 0.0;
    for (int j = threadIdx.x; j < D; j += blockDim.x) {
      T diff = X[i * D + j] - Y[i * D + j];
      dist += diff * diff;
    }

    float total_dist = BlockReduce(temp_storage).Sum(dist);
    __syncthreads();
    if (threadIdx.x == 0) {
      distance[i] = total_dist / 2.0;
    }
  }
}
}  // namespace

template<>
bool SquaredL2DistanceOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto* distance = Output(0);
  DCHECK_EQ(X.ndim(), Y.ndim());
  for (int i = 0; i < X.ndim(); ++i) {
    DCHECK_EQ(X.dim32(i), Y.dim32(i));
  }
  int N = X.ndim() > 0 ? X.dim32(0) : 1;
  int D = X.size() / N;
  distance->Resize(vector<TIndex>(size_t(1), N));
  SquaredL2DistanceKernel<<<
      std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N, D, X.data<float>(), Y.data<float>(), distance->mutable_data<float>());
  return true;
}

namespace {
template <typename T>
__global__ void
StripedScaleKernel(const int N, const int D, const T* alpha, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N * D) {
    int k = i / D;
    y[i] = x[i] * alpha[k];
  }
}
}

template <>
bool SquaredL2DistanceGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto& dDistance = Input(2);
  auto* dX = Output(0);
  auto* dY = Output(1);
  int N = X.ndim() > 0 ? X.dim32(0) : 1;
  int D = N > 0 ? X.size() / N : 0;
  CAFFE_ENFORCE(X.ndim() == Y.ndim());
  for (int i = 0; i < X.ndim(); ++i) {
    CAFFE_ENFORCE(X.dim32(i) == Y.dim32(i));
  }
  CAFFE_ENFORCE(dDistance.ndim() == 1);
  CAFFE_ENFORCE(dDistance.dim32(0) == N);
  dX->ResizeLike(X);
  dY->ResizeLike(Y);
  math::Sub<float, HIPContext>(
      X.size(),
      X.data<float>(),
      Y.data<float>(),
      dX->mutable_data<float>(),
      &context_);

  StripedScaleKernel<float><<<
      CAFFE_GET_BLOCKS(N * D),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N,
      D,
      dDistance.data<float>(),
      dX->data<float>(),
      dX->mutable_data<float>());

  // The gradient of the other side is basically the negative.
  math::Scale<float, HIPContext>(
      X.size(), -1, dX->data<float>(), dY->mutable_data<float>(), &context_);
  return true;
}

namespace {
REGISTER_HIP_OPERATOR(SquaredL2Distance,
                       SquaredL2DistanceOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SquaredL2DistanceGradient,
                       SquaredL2DistanceGradientOp<float, HIPContext>);
}  // namespace
}  // namespace caffe2
