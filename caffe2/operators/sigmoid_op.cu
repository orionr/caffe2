#include "hip/hip_runtime.h"
#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void SigmoidKernel(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = 1. / (1. + exp(-x[i]));
  }
}

template <typename T>
__global__ void SigmoidGradientKernel(const int N, const T* y, const T* dy,
                              T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = dy[i] * y[i] * (1. - y[i]);
  }
}

template <typename T>
struct SigmoidCUDAFunctor {
  inline void operator()(const int n, const float* x,
                         float* y, HIPContext* device_context) {
    SigmoidKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                    0, device_context->hip_stream()>>>(n, x, y);
    return;
  }
  inline bool InplaceAllowed() {
    return true;
  }
};

template <typename T>
struct SigmoidGradientCUDAFunctor {
  inline void operator()(const int n, const T* y, const T* dy,
                         T* dx, HIPContext* device_context) {
    SigmoidGradientKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                            0, device_context->hip_stream()>>>(n, y, dy, dx);
    return;
  }
  inline bool InplaceAllowed(const int input_id, const int output_id) {
    if (input_id == 1 && output_id == 0) {
      return true;
    } else {
      return false;
    }
  }
};

namespace {
REGISTER_HIP_OPERATOR(
    Sigmoid, UnaryElementwiseOp<float, HIPContext, SigmoidCUDAFunctor<float> >);
REGISTER_HIP_OPERATOR(
    SigmoidGradient, BinaryElementwiseOp<float, HIPContext,
                                     SigmoidGradientCUDAFunctor<float> >);
}  // namespace
}  // namespace caffe2