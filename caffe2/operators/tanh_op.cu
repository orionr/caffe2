#include "hip/hip_runtime.h"
#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void TanhKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = tanh(X[i]);
  }
}

template <typename T>
__global__ void TanhGradientKernel(const int N, const T* Y, const T* dY,
                              T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = dY[i]*(1 - Y[i]*Y[i]);
  }
}

template <typename T>
struct TanhCUDAFunctor {
  inline void operator()(const int n, const float* x,
                         float* y, HIPContext* device_context) {
    TanhKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                    0, device_context->hip_stream()>>>(n, x, y);
    return;
  }
  inline bool InplaceAllowed() {
    return true;
  }
};

template <typename T>
struct TanhGradientCUDAFunctor {
  inline void operator()(const int n, const T* y, const T* dy,
                         T* dx, HIPContext* device_context) {
    TanhGradientKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                            0, device_context->hip_stream()>>>(n, y, dy, dx);
    return;
  }
  inline bool InplaceAllowed(const int input_id, const int output_id) {
    if (input_id == 1 && output_id == 0) {
      return true;
    } else {
      return false;
    }
  }
};

namespace {
REGISTER_HIP_OPERATOR(
    Tanh, UnaryElementwiseOp<float, HIPContext, TanhCUDAFunctor<float> >);
REGISTER_HIP_OPERATOR(
    TanhGradient, BinaryElementwiseOp<float, HIPContext,
                                     TanhGradientCUDAFunctor<float> >);
}  // namespace
}  // namespace caffe2
