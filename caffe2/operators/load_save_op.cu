#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/load_save_op.h"

namespace caffe2 {
namespace {
REGISTER_HIP_OPERATOR(LoadTensor, LoadTensorOp<HIPContext>);
REGISTER_HIP_OPERATOR(Save, SaveOp<HIPContext>);
REGISTER_HIP_OPERATOR(Snapshot, SnapshotOp<HIPContext>);
}  // namespace
}  // namespace caffe2
