#include "hip/hip_runtime.h"
#include <math.h>
#include <cfloat>
// TODO(jamesreed): I would use <cmath> here but std::isnan
// and std::isinf are declared constexpr there and the nvidia
// compiler throws an error because of it

#include "caffe2/core/context_gpu.h"
#include "utility_ops.h"

namespace caffe2 {

__global__ void NanCheckKernel(int N, const float* X, bool* result) {
  bool has_nan = false;
  CUDA_1D_KERNEL_LOOP(i, N) {
    // Note: we have no need to do early return, since only if this fails
    // will we not need to inspect all elements. No need to optimize the
    // case that will fail.
    has_nan = has_nan || isnan(X[i]) || isinf(X[i]);
  }
  __syncthreads();
  if (has_nan) {
    result[0] = true;
  }
}

template <>
bool NanCheckOp<HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  const size_t N = X.size();
  const float* data_ptr = X.data<float>();

  scratch_.Resize(1);
  math::Set<bool, HIPContext>(
      1, false, scratch_.mutable_data<bool>(), &context_);
  NanCheckKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N, X.data<float>(), scratch_.mutable_data<bool>());

  bool result = false;
  {
    std::lock_guard<std::mutex> lock(HIPContext::mutex());
    CUDA_ENFORCE(hipMemcpyAsync(
        &result,
        scratch_.raw_data(),
        1,
        hipMemcpyDefault,
        context_.hip_stream()));
  }
  // Note: we must synchronize here so we can inspect the result
  context_.FinishDeviceComputation();

  // Print out diagnostic info if we have a NaN or inf
  if (result) {
    std::cerr << "Tensor contained NaN or inf: " << this->def().input(0)
              << std::endl;

    for (int j = 0; j < InputSize(); j++) {
      TensorCPU cpu_X;
      cpu_X.ResizeLike(Input(j));
      // Hack to cause allocaiton happen here, so it won't happen
      // when we do CopyFrom. We need the mutex then because host->gpu
      // copies seem to possibly lock with NCCL.
      cpu_X.mutable_data<float>();

      {
        std::lock_guard<std::mutex> lock(HIPContext::mutex());
        cpu_X.CopyFrom(Input(j), &context_);
      }
      context_.FinishDeviceComputation();
      std::cerr << "Input tensor: " << j << ": [" << def().input(j) << "]"
                << std::endl;
      tensorPrinter_.Print<float>(cpu_X);

      if (j == 0) {
        std::cerr << "NaN idxs:" << std::endl;
        auto* cpu_X_data = cpu_X.data<float>();
        for (size_t i = 0; i < cpu_X.size(); ++i) {
          if (isnan(cpu_X_data[i]) || isinf(cpu_X_data[i])) {
            std::cerr << i << " ";
          }
        }
      }
      std::cerr << std::endl;
    }
    return false;
  }

  // This op should act as an identity matrix if we don't find any NaNs/infs.
  // Copy over the data if we are not doing this in-place.
  if (&X != Y) {
    Y->CopyFrom(X, &context_);
  }
  return true;
}

REGISTER_HIP_OPERATOR(NanCheck, NanCheckOp<HIPContext>);

__global__ void
ElwiseMaxKernel(const float* X, const float* Y, float* maxout, const int N) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    maxout[i] = max(X[i], Y[i]);
  }
}

template <>
bool MaxOp<float, HIPContext>::Compute() {
  float* output_data = Output(0)->mutable_data<float>();
  const int N = Input(0).size();

  // Run pairwise-maxes
  for (int i = 1; i < InputSize(); ++i) {
    ElwiseMaxKernel<<<
        CAFFE_GET_BLOCKS(N),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        (i == 0 ? Input(0).data<float>() : Output(0)->data<float>()),
        Input(i).data<float>(),
        output_data,
        N);
  }

  return true;
}

REGISTER_HIP_OPERATOR(Max, MaxOp<float, HIPContext>);

template<typename T_INDEX>
__global__ void
GatherKernel(const float* X, float* Y, const T_INDEX* indices, const int N, const int block_size) {
  for (int i = blockIdx.x; i < N; i += gridDim.x) {
    T_INDEX idx = indices[i];
    const float* src_offset = X + idx * block_size;
    float* dst_offset = Y + i   * block_size;
    for (int j = threadIdx.x; j < block_size; j += blockDim.x) {
      dst_offset[j] = src_offset[j];
    }
  }
}

template <>
bool GatherOp<HIPContext>::RunOnDevice() {
  return DispatchHelper<TensorTypes<int32_t,int64_t>>::call(
      this, OperatorBase::Input<TensorCUDA>(INDICES));
}

template <>
template <typename Index>
bool GatherOp<HIPContext>::DoRunWithType() {
  auto& data = Input(DATA);
  auto& indices = Input(INDICES);
  auto* output = Output(0);

  CAFFE_ENFORCE_GE(data.ndim(), 1, "DATA should be at least 1-D");
  auto shape = indices.dims();
  shape.insert(shape.end(), data.dims().begin() + 1, data.dims().end());
  output->Resize(shape);

  int block_size = data.size() / data.dim(0);
  auto block_bytesize = data.size_from_dim(1) * data.meta().itemsize();
  CAFFE_ENFORCE(
      block_bytesize == data.nbytes() / data.dim(0),
      "block_bytesize should be consistent with data dim");
  int N = indices.size();

  auto src_base = static_cast<const float*>(data.raw_data());
  const Index* idxs = indices.template data<Index>();
  auto out = static_cast<float*>(output->raw_mutable_data(data.meta()));

  GatherKernel<<<
      std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
        src_base, out, idxs, N, block_size
      );
  return true;
}

namespace {
REGISTER_HIP_OPERATOR(Gather, GatherOp<HIPContext>);
}

}  // namespace caffe2
