#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/distance_op.h"

namespace caffe2 {

namespace {
// TODO(Yangqing): This function does very aweful memory access.
// Need improvement.
template <typename T>
__global__ void SquaredL2DistanceKernel(
    const int N, const int D, const T* X, const T* Y, T* distance) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    distance[i] = 0;
    for (int j = 0; j < D; ++j) {
      T diff = X[i * D + j] - Y[i * D + j];
      distance[i] += diff * diff;
    }
    distance[i] /= 2;
  }
}
}  // namespace

template<>
bool SquaredL2DistanceOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto* distance = Output(0);
  DCHECK_EQ(X.ndim(), Y.ndim());
  for (int i = 0; i < X.ndim(); ++i) {
    DCHECK_EQ(X.dim32(i), Y.dim32(i));
  }
  int N = X.ndim() > 0 ? X.dim32(0) : 1;
  int D = X.size() / N;
  distance->Resize(vector<TIndex>(size_t(1), N));
  SquaredL2DistanceKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                            0, context_.hip_stream()>>>(
      N, D, X.data<float>(), Y.data<float>(), distance->mutable_data<float>());
  return true;
}


namespace {
REGISTER_HIP_OPERATOR(SquaredL2Distance,
                       SquaredL2DistanceOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SquaredL2DistanceGradient,
                       SquaredL2DistanceGradientOp<float, HIPContext>);
}  // namespace
}  // namespace caffe2
