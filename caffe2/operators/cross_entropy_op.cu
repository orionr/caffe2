#include "hip/hip_runtime.h"
#include <assert.h>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/cross_entropy_op.h"
#include "caffe2/operators/operator_fallback_gpu.h"

namespace caffe2 {

namespace {
__global__ void LabelCrossEntropyKernel(
    const int N, const int D, const float* Xdata, const int* labeldata,
    const float log_threshold, float* Ydata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    CUDA_KERNEL_ASSERT(labeldata[i] >= 0 && labeldata[i] < D);
    Ydata[i] = -logf(max(Xdata[i * D + labeldata[i]], log_threshold));
  }
}
__global__ void LabelCrossEntropyGradientKernel(
    const int N, const int D, const float* Xdata, const int* labeldata,
    const float* dYdata, const float log_threshold, float* dXdata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    int idx = i * D + labeldata[i];
    dXdata[idx] = - dYdata[i] / max(Xdata[idx], log_threshold);
  }
}
}  // namespace

template <>
bool LabelCrossEntropyOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& label = Input(1);
  auto* Y = Output(0);
  DCHECK_EQ(X.ndim(), 2);
  int N = X.dim32(0);
  int D = X.dim32(1);
  DCHECK((label.ndim() == 1) || (label.ndim() == 2 && label.dim32(1) == 1));
  DCHECK_EQ(label.dim32(0), N);
  Y->Resize(vector<TIndex>(size_t(1), N));
  LabelCrossEntropyKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                            0, context_.hip_stream()>>>(
      N, D, X.data<float>(), label.data<int>(), kLOG_THRESHOLD(),
      Y->mutable_data<float>());
  return true;
}

template <>
bool LabelCrossEntropyGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& label = Input(1);
  auto& dY = Input(2);
  auto* dX = Output(0);
  DCHECK_EQ(X.ndim(), 2);
  int N = X.dim32(0);
  int D = X.dim32(1);
  DCHECK((label.ndim() == 1) || (label.ndim() == 2 && label.dim32(1) == 1));
  DCHECK_EQ(label.dim32(0), N);
  DCHECK_EQ(dY.ndim(), 1);
  DCHECK_EQ(dY.dim32(0), N);
  dX->ResizeLike(X);
  math::Set<float, HIPContext>(
      dX->size(), 0.f, dX->mutable_data<float>(), &context_);
  LabelCrossEntropyGradientKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                                    0, context_.hip_stream()>>>(
      N, D, X.data<float>(), label.data<int>(), dY.data<float>(),
      kLOG_THRESHOLD(), dX->mutable_data<float>());
  return true;
}

namespace {
__global__ void MakeTwoClassKernel(
    const int N, const float* Xdata, float* Ydata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Ydata[i * 2] = 1.0 - Xdata[i];
    Ydata[i * 2 + 1] = Xdata[i];
  }
}
__global__ void MakeTwoClassGradientKernel(
    const int N, const float* dYdata, float* dXdata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dXdata[i] = dYdata[i * 2 + 1] - dYdata[i * 2];
  }
}
}  // namespace

template <>
bool MakeTwoClassOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  auto shape = X.dims();
  shape.push_back(2);
  CAFFE_ENFORCE_LT(X.size(), std::numeric_limits<int>::max() / 2);
  Y->Resize(shape);
  int N = X.size();
  MakeTwoClassKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                       0, context_.hip_stream()>>>(
      N, X.data<float>(), Y->mutable_data<float>());
  return true;
}

template <>
bool MakeTwoClassGradientOp<float, HIPContext>::RunOnDevice() {
  auto& dY = Input(0);
  auto* dX = Output(0);
  auto shape = dY.dims();
  CAFFE_ENFORCE_GE(shape.size(), 1);
  CAFFE_ENFORCE_EQ(shape.back(), 2);
  shape.pop_back();
  CAFFE_ENFORCE_LT(dY.size(), std::numeric_limits<int>::max());
  dX->Resize(shape);
  int N = dX->size();
  MakeTwoClassGradientKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                               0, context_.hip_stream()>>>(
      N, dY.data<float>(), dX->mutable_data<float>());
  return true;
}

namespace {
REGISTER_HIP_OPERATOR(LabelCrossEntropy,
                       LabelCrossEntropyOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(LabelCrossEntropyGradient,
                       LabelCrossEntropyGradientOp<float, HIPContext>);

REGISTER_HIP_OPERATOR(MakeTwoClass,
                       MakeTwoClassOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(MakeTwoClassGradient,
                       MakeTwoClassGradientOp<float, HIPContext>);

//TODO(surya) Add full GPU/CUDA support for the CrossEntropyOp
REGISTER_HIP_OPERATOR(CrossEntropy,
                       GPUFallbackOp<CrossEntropyOp<float, CPUContext>>);
REGISTER_HIP_OPERATOR(CrossEntropyGradient,
                       GPUFallbackOp<CrossEntropyGradientOp<float, CPUContext>>);

}  // namespace
}  // namespace caffe2
