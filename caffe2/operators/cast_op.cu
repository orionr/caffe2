#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/cast_op.h"

namespace caffe2 {

namespace {
template <typename DstType, typename SrcType>
__global__ void CastKernel(const int N, const SrcType* X, DstType* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = static_cast<DstType>(X[i]);
  }
}
}  // namespace

template <>
template <typename DstType, typename SrcType>
bool CastOp<HIPContext>::DoRunWithType() {
  auto& input = Input(0);
  auto* output = Output(0);
  output->ResizeLike(input);
  const auto* data = input.template data<SrcType>();
  auto* out = output->template mutable_data<DstType>();
  DCHECK(input.size() < INT_MAX);
  int N = input.size();
  CastKernel<DstType, SrcType><<<
      CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
      0, context_.hip_stream()>>>(N, data, out);
  return true;
}

REGISTER_HIP_OPERATOR(Cast, CastOp<HIPContext>);

}  // namespace caffe2
