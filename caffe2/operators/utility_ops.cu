#include "hip/hip_runtime.h"
#include <math.h>
#include <cfloat>
// TODO(jamesreed): I would use <cmath> here but std::isnan
// and std::isinf are declared constexpr there and the nvidia
// compiler throws an error because of it

#include "caffe2/core/context_gpu.h"
#include "utility_ops.h"

namespace caffe2 {
CAFFE_KNOWN_TYPE(const float*);

__global__ void NanCheckKernel(int N, const float* X, bool* result) {
  bool has_nan = false;
  CUDA_1D_KERNEL_LOOP(i, N) {
    // Note: we have no need to do early return, since only if this fails
    // will we not need to inspect all elements. No need to optimize the
    // case that will fail.
    has_nan = has_nan || isnan(X[i]) || isinf(X[i]);
  }
  __syncthreads();
  if (has_nan) {
    result[0] = true;
  }
}

template <>
bool NanCheckOp<HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  const size_t N = X.size();
  const float* data_ptr = X.data<float>();

  scratch_.Resize(1);
  math::Set<bool, HIPContext>(
      1, false, scratch_.mutable_data<bool>(), &context_);
  NanCheckKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N, X.data<float>(), scratch_.mutable_data<bool>());

  bool result = false;
  {
    std::lock_guard<std::mutex> lock(HIPContext::mutex());
    CUDA_ENFORCE(hipMemcpyAsync(
        &result,
        scratch_.raw_data(),
        1,
        hipMemcpyDefault,
        context_.hip_stream()));
  }
  // Note: we must synchronize here so we can inspect the result
  context_.FinishDeviceComputation();

  // Print out diagnostic info if we have a NaN or inf
  if (result) {
    std::cerr << "Tensor contained NaN or inf: " << this->def().input(0)
              << std::endl;

    for (int j = 0; j < InputSize(); j++) {
      TensorCPU cpu_X;
      cpu_X.ResizeLike(Input(j));
      // Hack to cause allocaiton happen here, so it won't happen
      // when we do CopyFrom. We need the mutex then because host->gpu
      // copies seem to possibly lock with NCCL.
      cpu_X.mutable_data<float>();

      {
        std::lock_guard<std::mutex> lock(HIPContext::mutex());
        cpu_X.CopyFrom(Input(j), &context_);
      }
      context_.FinishDeviceComputation();
      std::cerr << "Input tensor: " << j << ": [" << def().input(j) << "]"
                << std::endl;
      tensorPrinter_.Print<float>(cpu_X);

      if (j == 0) {
        std::cerr << "NaN idxs:" << std::endl;
        auto* cpu_X_data = cpu_X.data<float>();
        for (size_t i = 0; i < cpu_X.size(); ++i) {
          if (isnan(cpu_X_data[i]) || isinf(cpu_X_data[i])) {
            std::cerr << i << " ";
          }
        }
      }
      std::cerr << std::endl;
    }
    return false;
  }

  // This op should act as an identity matrix if we don't find any NaNs/infs.
  // Copy over the data if we are not doing this in-place.
  if (&X != Y) {
    Y->CopyFrom(X, &context_);
  }
  return true;
}

REGISTER_HIP_OPERATOR(NanCheck, NanCheckOp<HIPContext>);

__global__ void
ElwiseMaxKernel(const float* X, const float* Y, float* maxout, const int N) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    maxout[i] = max(X[i], Y[i]);
  }
}

template <>
bool MaxOp<float, HIPContext>::Compute() {
  float* output_data = Output(0)->mutable_data<float>();
  const int N = Input(0).size();

  // Run pairwise-maxes
  for (int i = 1; i < InputSize(); ++i) {
    ElwiseMaxKernel<<<
        CAFFE_GET_BLOCKS(N),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        (i == 0 ? Input(0).data<float>() : Output(0)->data<float>()),
        Input(i).data<float>(),
        output_data,
        N);
  }

  return true;
}

REGISTER_HIP_OPERATOR(Max, MaxOp<float, HIPContext>);

template<typename T_INDEX>
__global__ void
GatherKernel(const float* X, float* Y, const T_INDEX* indices, const int N, const int block_size) {
  for (int i = blockIdx.x; i < N; i += gridDim.x) {
    T_INDEX idx = indices[i];
    const float* src_offset = X + idx * block_size;
    float* dst_offset = Y + i   * block_size;
    for (int j = threadIdx.x; j < block_size; j += blockDim.x) {
      dst_offset[j] = src_offset[j];
    }
  }
}

template <>
bool GatherOp<HIPContext>::RunOnDevice() {
  return DispatchHelper<TensorTypes<int32_t,int64_t>>::call(
      this, OperatorBase::Input<TensorCUDA>(INDICES));
}

template <>
template <typename Index>
bool GatherOp<HIPContext>::DoRunWithType() {
  auto& data = Input(DATA);
  auto& indices = Input(INDICES);
  auto* output = Output(0);

  CAFFE_ENFORCE_GE(data.ndim(), 1, "DATA should be at least 1-D");
  auto shape = indices.dims();
  shape.insert(shape.end(), data.dims().begin() + 1, data.dims().end());
  output->Resize(shape);

  int block_size = data.size() / data.dim(0);
  auto block_bytesize = data.size_from_dim(1) * data.meta().itemsize();
  CAFFE_ENFORCE(
      block_bytesize == data.nbytes() / data.dim(0),
      "block_bytesize should be consistent with data dim");
  int N = indices.size();

  auto src_base = static_cast<const float*>(data.raw_data());
  const Index* idxs = indices.template data<Index>();
  auto out = static_cast<float*>(output->raw_mutable_data(data.meta()));

  GatherKernel<<<
      std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
        src_base, out, idxs, N, block_size
      );
  return true;
}

REGISTER_HIP_OPERATOR(Gather, GatherOp<HIPContext>);

/**
 * @brief Update slices of Y in-place with a batch of weighted X's.
 * Y[idx] = alpha[b] * X[b][i] + Y[idx]
 * i=0,...,N-1
 * b=0,...,B-1
 * idx=Indices[i]
 */
template<typename T_INDEX>
__global__ void 
AxpySliceKernel(
             const TIndex N,
             const TIndex B,
             const TIndex slice_size,
             const float** alpha,
             const float** X,
             const T_INDEX* Indices, 
             float* Y,
             const TIndex M) {
  for (int i = blockIdx.x; i < N; i += gridDim.x) {
    T_INDEX idx = Indices[i];
    float* y_offset = Y + (idx * slice_size);
    for (int b = 0; b < B; b++) {
      const float* x_offset = X[b] + (i * slice_size);
      for (int j = threadIdx.x; j < slice_size; j += blockDim.x) {
        atomicAdd(&y_offset[j], (*alpha[b]) * x_offset[j]);
      }
    }
  }
}

template <>
bool ScatterWeightedSumOp<float,HIPContext>::RunOnDevice() {
    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(this, Input(2));
}

template <>
template <typename Index>
bool ScatterWeightedSumOp<float,HIPContext>::DoRunWithType() {
  DCHECK_EQ(InputSize() % 2, 1);
  auto& X0 = Input(0);
  auto& weight0 = Input(1);
  auto& indices = Input(2);
  auto* output = Output(0);

  CAFFE_ENFORCE_EQ(&X0, output, "In place operation is required");
  DCHECK_GT(X0.size(), 0);
  DCHECK_GT(X0.ndim(), 0) << "X0 has to be at least the vector";
  DCHECK_EQ(weight0.size(), 1);

  TIndex M = X0.size();
  TIndex N = X0.dim(0);
  TIndex K = indices.size();
  TIndex block_size = M / N;

  T* data = output->template mutable_data<T>();
  const Index* Indices = indices.template data<Index>();

  float w0;
  context_.Copy<float,HIPContext,CPUContext>(1, weight0.template data<float>(), &w0);
  OPERATOR_NEEDS_FEATURE(
    w0 == 1.0,
    "ScatterWeightedSumOp only supports weight_0=1 on HIPContext");

  const TIndex B = (InputSize()-3)/2;

  // In order to have all device pointers of x_i (and weight_i similarly) consecutively 
  // in device memory, copy pointers to a host vector and then copy back into a device array.
  x_data_host_.Resize(B);
  weights_host_.Resize(B);
  x_data_device_.Resize(B);
  weights_device_.Resize(B);
  CAFFE_ENFORCE(x_data_host_.size() == B); 
  CAFFE_ENFORCE(weights_host_.size() == B); 
  CAFFE_ENFORCE(x_data_device_.size() == B); 
  CAFFE_ENFORCE(weights_device_.size() == B); 
  const float** x_data_host = x_data_host_.mutable_data<const float*>();
  const float** weights_host = weights_host_.mutable_data<const float*>();
  const float** x_data_device = x_data_device_.mutable_data<const float*>();
  const float** weights_device = weights_device_.mutable_data<const float*>();
  for (int inp = 3; inp < InputSize(); inp += 2) {
    x_data_host [(inp-3)/2] = static_cast<const float*>(Input(inp).raw_data());
    weights_host[(inp-3)/2] = static_cast<const float*>(Input(inp+1).raw_data());
  }
  context_.Copy<const float*,CPUContext,HIPContext>(B, x_data_host, x_data_device);
  context_.Copy<const float*,CPUContext,HIPContext>(B, weights_host, weights_device);

  AxpySliceKernel<<<
    std::min<TIndex>(K, CAFFE_MAXIMUM_NUM_BLOCKS),
    CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>
    (
      K, B, block_size, weights_device, x_data_device, Indices, data, M
    );

  return true;
}

REGISTER_HIP_OPERATOR(ScatterWeightedSum, ScatterWeightedSumOp<float,HIPContext>);

}  // namespace caffe2
