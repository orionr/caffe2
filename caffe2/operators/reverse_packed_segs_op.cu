#include "caffe2/core/context_gpu.h"
#include "reverse_packed_segs_op.h"

namespace caffe2 {
namespace {
REGISTER_HIP_OPERATOR(ReversePackedSegs, ReversePackedSegsOp<HIPContext>);

} // namespace
} // namespace caffe2
