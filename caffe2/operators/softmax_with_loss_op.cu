#include "hip/hip_runtime.h"
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "softmax_with_loss_op.h"

namespace caffe2 {

namespace {

__global__ void LabelCrossEntropyKernel(
    const int N, const int D, const float* Pdata, const int* labeldata,
    const float* weights, float* Ydata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    CUDA_KERNEL_ASSERT(labeldata[i] < D);
    float weight = weights ? weights[i] : 1.0;
    Ydata[i] = -logf(max(Pdata[i * D + labeldata[i]], FLT_MIN)) * weight;
  }
}

__global__ void LabelCrossEntropyGradientKernel(
    const int N, const int D, const float* Pdata, const int* labeldata,
    float* dXdata, const float *weights) {
      if (weights == NULL) {
        CUDA_1D_KERNEL_LOOP(i, N) {
         int idx = i * D + labeldata[i];
         dXdata[idx] = Pdata[idx] - 1.;
       }
     } else {
       CUDA_1D_KERNEL_LOOP(i, N) {
        int idx = i * D + labeldata[i];
        dXdata[idx] = Pdata[idx] - 1.;

        float weight = weights[i];
        for(int d=0; d<D; d++) {
            int idx = i * D + d;
            dXdata[idx] *= weight;
        }
     }
   }
}

__global__ void RowMaxKernel(const int num, const int D, const float* data,
    float* out) {
  CUDA_1D_KERNEL_LOOP(index, num) {
    float maxval = -FLT_MAX;
    for (int d = 0; d < D; ++d) {
      maxval = max(data[index * D + d], maxval);
    }
    out[index] = maxval;
  }
}


__global__ void SpatialSoftmaxKernel(const int num, const int D, const int W, const int H,
      const float* Xdata, float* Pdata) {
  CUDA_1D_KERNEL_LOOP(i, num) {
    for(int y = 0; y < H; ++y) {
      for(int x = 0; x < W; ++x) {
        // Subtract max on each cell for numerical reasons
        float max_val = -FLT_MAX;
        for(int c = 0; c < D; ++c) {
          // TODO optimize
          int idx = i * (H * W * D) + c * (H * W) + y * W + x;
          max_val = max(max_val, Xdata[idx]);
        }

        // Exponentiate
        float expsum = 0.0f;
        for(int c = 0; c < D; ++c) {
          int idx = i * (H * W * D) + c * (H * W) + y * W + x;
          float expx = exp(Xdata[idx] - max_val);
          Pdata[idx] = expx;
          expsum += expx;
        }

        // Normalize
        for(int c=0; c<D; ++c) {
          int idx = i * (H * W * D) + c * (H * W) + y * W + x;
          Pdata[idx] /= expsum;
        }
      }
    }
  }
}

#define DONTCARE (-1)

#define REDUCTION_KERNEL_THREADS_X 16
#define REDUCTION_KERNEL_THREADS_Y 16
#define REDUCTION_THREADS (REDUCTION_KERNEL_THREADS_X * REDUCTION_KERNEL_THREADS_Y)

__global__ void SpatialCrossEntropyLossKernel(const int N, const int D, const int W, const int H,
    const float* Pdata, const int* label_data, const float *weights,
      float* avg_loss_data, float *total_weight_ret) {
    __shared__ float sum_buf[REDUCTION_THREADS];
    __shared__ float total_weight_buffer[REDUCTION_THREADS];

    const int thread_idx = REDUCTION_KERNEL_THREADS_X * threadIdx.y + threadIdx.x;
    float sum_label_xent = 0.0;
    float total_weight = 0.0f;
    for (int x = (blockIdx.x * blockDim.x) + threadIdx.x;
             x < W;
             x += blockDim.x * gridDim.x) {
      for (int y = (blockIdx.y * blockDim.y) + threadIdx.y;
               y < H;
               y += blockDim.y * gridDim.y) {
        for(int i = 0; i < N; ++i) {
          int labelidx =  i * H * W + y * W + x;
          int label = label_data[labelidx];
          if (label != DONTCARE) {
            float weight = (weights == NULL ? 1.0 : weights[labelidx]);
            int idx = i * (H * W * D) + label * (H * W) + y * W + x;
            sum_label_xent += -logf(max(Pdata[idx], 1e-20f)) * weight;
            total_weight += weight;
          }
        }

      }
    }
    sum_buf[thread_idx] = sum_label_xent;
    total_weight_buffer[thread_idx] = total_weight;

    __syncthreads();

    if (thread_idx == 0) {
      // TODO: multi-level reduction
      float sum_xent = 0;
      float sum_total_weight = 0.0f;
      for(int j = 0; j < REDUCTION_THREADS; ++j) {
        sum_xent += sum_buf[j];
        sum_total_weight += total_weight_buffer[j];
      }

      *avg_loss_data = (*avg_loss_data) + sum_xent;
      *total_weight_ret = (*total_weight_ret) + sum_total_weight;
    }

    __syncthreads();
  }

__global__ void SpatialSoftmaxLossGradientKernel(const int N, const int D,
    const int W, const int H, const int* label_data, const float* weights,
         float* dX_data, float* total_weight_ret) {
      __shared__ float total_weight_buffer[REDUCTION_THREADS];

      const int thread_idx = REDUCTION_KERNEL_THREADS_X * threadIdx.y + threadIdx.x;

      float total_weight = 0.0;
      for (int x = (blockIdx.x * blockDim.x) + threadIdx.x;
               x < W;
               x += blockDim.x * gridDim.x) {
        for (int y = (blockIdx.y * blockDim.y) + threadIdx.y;
                 y < H;
                 y += blockDim.y * gridDim.y) {
           for (int i = 0; i < N; ++i) {
             int labelidx = i * H * W + y * W + x;
             int label = label_data[labelidx];
             if (label != DONTCARE) {
               int idx = i * (H * W * D) + label * (H * W) + y * W + x;
               dX_data[idx] = (dX_data[idx] - 1.0);

               if (weights != NULL) {
                 float weight = weights[labelidx];
                 for (int c = 0; c < D; ++c) {
                   int idx = i * (H * W * D) + c * (H * W) + y * W + x;
                   dX_data[idx] *= weight;
                 }
                 total_weight += weight;
               } else {
                 total_weight += 1.0;
               }
             } else {
               // Ignore-label, so set all gradients for this positions
               // tp zero
               for (int c = 0; c < D; ++c) {
                 int idx = i * (H * W * D) + c * (H * W) + y * W + x;
                 dX_data[idx] = 0.0;
               }
             }
           }
         }
       }
       total_weight_buffer[thread_idx] = total_weight;
       __syncthreads();

       if (thread_idx == 0) {
         // TODO: multi-level reduction
         float sum_total_weight = 0.0f;
         for(int j = 0; j < REDUCTION_THREADS; ++j) {
           sum_total_weight += total_weight_buffer[j];
         }
         *total_weight_ret = (*total_weight_ret) + sum_total_weight;
       }

       __syncthreads();
    }

__global__ void SoftmaxNormalizeKernel(
    const int nthreads, const int D, const float* Pdata, const float* scales,
    float* out) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index / D;
    out[index] = Pdata[index] / scales[n];
  }
}

void Softmax(const int N, const int D, const float* logits, const int* labels,
             const float* sum_multiplier, float* scales, float* probs,
             HIPContext* context) {
  const int size = N * D;
  RowMaxKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                 0, context->hip_stream()>>>(N, D, logits, scales);
  // Put the intermediate result X - max(X) into Y
  context->Copy<float, HIPContext, HIPContext>(size, logits, probs);
  // Subtract the scale
  math::Gemm<float, HIPContext>(CblasNoTrans, CblasNoTrans, N, D, 1,
                                 -1, scales, sum_multiplier, 1, probs, context);
  // Exponentiation
  math::Exp<float, HIPContext>(size, probs, probs, context);
  // Sum exponentiated values
  math::Gemv<float, HIPContext>(CblasNoTrans, N, D, 1, probs, sum_multiplier,
                                 0, scales, context);
  // Normalize
  SoftmaxNormalizeKernel<<<CAFFE_GET_BLOCKS(size), CAFFE_CUDA_NUM_THREADS,
                           0, context->hip_stream()>>>(
    size, D, probs, scales, probs);
}

} // namespace

template<>
bool SoftmaxWithLossOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);  // Logits
  auto& T = Input(1);  // Labels / targets
  auto* P = Output(0); // Probabilities from softmax
  auto* avg_loss = Output(1); // Average loss
  const float* weights = (InputSize() > 2 ? Input(2).data<float>() : NULL);

  int N = X.dim32(0);
  int D = X.dim32(1);
  P->ResizeLike(X);

  if (!spatial_mode_) {
    DCHECK_EQ(X.ndim(), 2);
    DCHECK((T.ndim() == 1) || (T.ndim() == 2 && T.dim32(1) == 1));
    DCHECK_EQ(T.dim32(0), N);

    avg_loss->Resize(vector<TIndex>());
    if (losses_.size() != N) {
      losses_.Resize(N);
    }
    if (sum_multiplier_.size() != D) {
      sum_multiplier_.Resize(D);
      math::Set<float, HIPContext>(
          D, 1.f, sum_multiplier_.mutable_data<float>(), &context_);
    }
    Softmax(N, D, X.data<float>(), T.data<int>(), sum_multiplier_.data<float>(),
            losses_.mutable_data<float>(), P->mutable_data<float>(), &context_);
    // Compute label xent loss per example
    LabelCrossEntropyKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                              0, context_.hip_stream()>>>(
        N, D, P->data<float>(), T.data<int>(), weights,
        losses_.mutable_data<float>());

    float total_weight = N;
    if (weights) {
      // Sum weights
      float* total_weight_ptr;
      hipMalloc(&total_weight_ptr, sizeof(float));
      math::Sum<float, HIPContext>(N, weights, total_weight_ptr, &context_);
      hipMemcpyAsync(&total_weight, total_weight_ptr, sizeof(float),
        hipMemcpyDeviceToHost, context_.hip_stream());
      hipFree(total_weight_ptr);
    }

    // Sum of all losses
    float* avg_loss_data = avg_loss->mutable_data<float>();
    math::Sum<float, HIPContext>(
        losses_.size(), losses_.data<float>(), avg_loss_data, &context_);
    // Average of input batch size
    math::Scale<float, HIPContext>(
        1, scale_ / total_weight, avg_loss_data, avg_loss_data, &context_);
  } else {
    DCHECK_EQ(X.ndim(), 4);
    DCHECK_EQ(T.ndim(), 3);

    int H = X.dim32(2);
    int W = X.dim32(3);

    const float* Xdata = X.data<float>();
    float* Pdata = P->mutable_data<float>();

    // Softmax for each x,y location
    SpatialSoftmaxKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                           0, context_.hip_stream()>>>(
        N, D, W, H, Xdata, Pdata);

    // Cross entropy
    avg_loss->Resize(vector<TIndex>());
    float* avg_loss_data = avg_loss->mutable_data<float>();
    math::Set<float, HIPContext>(1, 0.0f, avg_loss_data, &context_);

    const int* label_data = T.data<int>();
    float* total_weight_ptr;
    hipMalloc(&total_weight_ptr, sizeof(float));
    math::Set<float, HIPContext>(1, 0.0f, total_weight_ptr, &context_);

    // TODO: how to set best?
    dim3 threadsPerBlock(REDUCTION_KERNEL_THREADS_X, REDUCTION_KERNEL_THREADS_Y);
    dim3 numBlocks(1, 1);
    SpatialCrossEntropyLossKernel<<<numBlocks, threadsPerBlock,
        0, context_.hip_stream()>>>(
        N, D, W, H, P->data<float>(), label_data, weights,
        avg_loss_data, total_weight_ptr);


    // Somewhat awkward scalar passing from device to host
    float h_total_weight;
    hipMemcpyAsync(&h_total_weight, total_weight_ptr, sizeof(float),
      hipMemcpyDeviceToHost, context_.hip_stream());
    hipFree(total_weight_ptr);

    // Final scaling
    math::Scale<float, HIPContext>(
        1, scale_ / h_total_weight,
        avg_loss_data, avg_loss_data, &context_);

  }
  return true;
}


template<>
bool SoftmaxWithLossGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);  // Logits
  auto& T = Input(1);  // Labels / targets
  // Input(2) is weights, if given
  auto& P = Input(InputSize() - 2);  // Probabilities from softmax
  auto& d_avg_loss = Input(InputSize() - 1); // Gradient w.r.t. avg loss
  const float* weights = (InputSize() > 4 ? Input(2).data<float>() : NULL);

  auto* dX = Output(0);
  int N = X.dim32(0);
  int D = X.dim32(1);
  dX->ResizeLike(X);

  if (!spatial_mode_) {
    DCHECK_EQ(X.ndim(), 2);
    DCHECK((T.ndim() == 1) || (T.ndim() == 2 && T.dim32(1) == 1));
    DCHECK_EQ(T.dim32(0), N);
    // Copy softmax probabilities into dX
    context_.Copy<float, HIPContext, HIPContext>(
        P.size(), P.data<float>(), dX->mutable_data<float>());
    // Subtract 1 from labeled positions
    LabelCrossEntropyGradientKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                                      0, context_.hip_stream()>>>(
        N, D, P.data<float>(), T.data<int>(), dX->mutable_data<float>(),
        weights);

    float total_weight = N;
    if (weights) {
      // Sum weights
      float* total_weight_ptr;
      hipMalloc(&total_weight_ptr, sizeof(float));
      math::Sum<float, HIPContext>(N, weights, total_weight_ptr, &context_);
      hipMemcpyAsync(&total_weight, total_weight_ptr, sizeof(float),
        hipMemcpyDeviceToHost, context_.hip_stream());
      hipFree(total_weight_ptr);
    }
        
    // Scale by d_avg_loss / N
    math::Scale<float, HIPContext>(
        dX->size(), scale_ / total_weight, dX->data<float>(),
        dX->mutable_data<float>(), &context_);
    math::Scale<float, HIPContext>(
        dX->size(), d_avg_loss.data<float>(), dX->data<float>(),
        dX->mutable_data<float>(), &context_);
  } else {
    // Spatial mode, compute softmax for each x, y location
    DCHECK_EQ(X.ndim(), 4);
    DCHECK_EQ(T.ndim(), 3);

    int H = X.dim32(2);
    int W = X.dim32(3);
    dX->ResizeLike(X);

    const float* Pdata = P.data<float>();
    float* dX_data = dX->mutable_data<float>();
    const int* label_data = T.data<int>();
    const float* d_avg_loss_data = d_avg_loss.data<float>();

    // Copy softmax probabilities into dX. All but the neuron
    // corresponding to the correct label has gradient equaling e(x_j)
    // which is the probability under softmax.
    context_.Copy<float, HIPContext, HIPContext>(P.size(), Pdata, dX_data);

    // TODO: how to set best?
    dim3 threadsPerBlock(REDUCTION_KERNEL_THREADS_X, REDUCTION_KERNEL_THREADS_Y);
    dim3 numBlocks(1, 1);

    float* total_weight_ptr;
    hipMalloc(&total_weight_ptr, sizeof(float));
    math::Set<float, HIPContext>(1, 0.0f, total_weight_ptr, &context_);

    SpatialSoftmaxLossGradientKernel<<<numBlocks, threadsPerBlock,
          0, context_.hip_stream()>>>(
        N, D, W, H, label_data, weights, dX_data,
        total_weight_ptr);

    // Somewhat awkward scalar passing from device to host
    float h_total_weight;
    hipMemcpyAsync(&h_total_weight, total_weight_ptr, sizeof(float),
      hipMemcpyDeviceToHost, context_.hip_stream());
    hipFree(total_weight_ptr);

    // Final scaling
    math::Scale<float, HIPContext>(
        dX->size(),
        scale_ / h_total_weight,
        dX->data<float>(),
        dX->mutable_data<float>(), &context_);
    math::Scale<float, HIPContext>(
        dX->size(),
        d_avg_loss.data<float>(),
        dX->data<float>(),
        dX->mutable_data<float>(), &context_);
  }
  return true;
}


namespace {
REGISTER_HIP_OPERATOR(SoftmaxWithLoss,
                       SoftmaxWithLossOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SoftmaxWithLossGradient,
                       SoftmaxWithLossGradientOp<float, HIPContext>);
} // namespace
} // namespace caffe2
