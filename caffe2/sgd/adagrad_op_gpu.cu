#include "hip/hip_runtime.h"
#include "adagrad_op.h"
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

__global__ void AdagradUpdate(
    int N,
    const float* g,
    const float* h,
    float* ng,
    float* nh,
    float epsilon,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float hi = nh[i] = h[i] + gi * gi;
    ng[i] = lr[0] * gi / (sqrt(hi) + epsilon);
  }
}

template <>
void adagrad_update<HIPContext>(
    int N,
    const float* g,
    const float* h,
    float* ng,
    float* nh,
    float epsilon,
    const float* lr,
    HIPContext* context) {
  AdagradUpdate<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, g, h, ng, nh, epsilon, lr);
}

namespace {
REGISTER_HIP_OPERATOR(Adagrad, AdagradOp<float, HIPContext>);
}
}
