#include "hip/hip_runtime.h"
#include <assert.h>

#include "elementwise_linear_op.h"

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/operator_fallback_gpu.h"


namespace caffe2 {

namespace {
__global__ void ElementwiseLinearKernel(const int N, const int D,
  const float* X_data, const float* a_data, const float* b_data,
  float* Y_data) {
    CUDA_1D_KERNEL_LOOP(i, N * D) {
      int d = i % D;
      Y_data[i] = X_data[i] * a_data[d] + b_data[d];
    }
}

__global__ void ElementwiseLinearGradientKernel(const int N, const int D,
  const float* g_o_data, const float* X_data, const float* a_data,
  float* g_X_data, float* g_a_data, float* g_b_data) {
  CUDA_1D_KERNEL_LOOP(d, D) {
    for (int n = 0; n < N; ++n) {
      g_X_data[n * D + d] = g_o_data[n * D + d] * a_data[d];
      g_a_data[d] += g_o_data[n * D + d] * X_data[n * D + d];
      g_b_data[d] += g_o_data[n * D + d];
    }
  }
}

}  // namespace


template<>
bool ElementwiseLinearOp<float, HIPContext>::RunOnDevice(){
  const auto& X = Input(0);
  const auto& a = Input(1);
  const auto& b = Input(2);
  auto* Y = Output(0);
  CAFFE_ENFORCE(X.ndim() == 2, X.ndim());
  CAFFE_ENFORCE(a.ndim() == 1, a.ndim());
  CAFFE_ENFORCE(X.dim32(1) == a.dim32(0));
  CAFFE_ENFORCE(a.dims() == b.dims());
  Y->ResizeLike(X);

  const int N = X.dim32(0);
  const int D = X.dim32(1);

  ElementwiseLinearKernel<<<CAFFE_GET_BLOCKS(N * D), CAFFE_CUDA_NUM_THREADS,
                          0, context_.hip_stream()>>>(
    N, D, X.data<float>(), a.data<float>(), b.data<float>(),
    Y->mutable_data<float>());
  return true;
}


template<>
bool ElementwiseLinearGradientOp<float, HIPContext>::RunOnDevice(){
  const auto& g_o = Input(0);
  const auto& X = Input(1);
  const auto& a = Input(2);
  CAFFE_ENFORCE(X.ndim() == 2, X.ndim());
  CAFFE_ENFORCE(a.ndim() == 1, a.ndim());
  CAFFE_ENFORCE(X.dim32(1) == a.dim32(0));

  auto *g_X = Output(0);
  auto *g_a = Output(1);
  auto *g_b = Output(2);
  g_X->ResizeLike(X);
  g_a->ResizeLike(a);
  g_b->ResizeLike(a);

  const int N = X.dim32(0);
  const int D = X.dim32(1);

  float* g_a_data = g_a->mutable_data<float>();
  float* g_b_data = g_b->mutable_data<float>();
  math::Set<float, HIPContext>(g_a->size(), 0.f, g_a_data, &context_);
  math::Set<float, HIPContext>(g_b->size(), 0.f, g_b_data, &context_);

  ElementwiseLinearGradientKernel<<<CAFFE_GET_BLOCKS(D), CAFFE_CUDA_NUM_THREADS,
                                  0, context_.hip_stream()>>>(
    N, D, g_o.data<float>(), X.data<float>(), a.data<float>(),
    g_X->mutable_data<float>(), g_a_data, g_b_data);
  return true;
}

namespace {

REGISTER_HIP_OPERATOR(ElementwiseLinear,
                       ElementwiseLinearOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(ElementwiseLinearGradient,
                       ElementwiseLinearGradientOp<float, HIPContext>);

}  // namespace

}  // namespace caffe2
